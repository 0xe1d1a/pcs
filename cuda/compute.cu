#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <math.h>
#include <stdlib.h>
#include <float.h>
#include <stdio.h>
extern "C" {
#include "compute.h"
}

#define CUDA_SAFE_CALL(ret) { gpuAssert((ret), __FILE__, __LINE__); }

// http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/* Does the reduction step and return if the convergence has setteled */
static int fill_report(const struct parameters *p, struct results *r,
                        size_t h, size_t w, 
                        double * a,
                        double * b,
                        double iter,
                        struct timeval *before)
{
    /* compute min/max/avg */
    double tmin = DBL_MAX, tmax = DBL_MIN;
    double sum = 0.0;
    double maxdiff = 0.0;
    struct timeval after;

    /* We have said that the final reduction does not need to be included. */
    gettimeofday(&after, NULL);
 
    for (size_t i = 1; i < h - 1; ++i)
        for (size_t j = 1; j < w - 1; ++j) 
        {
            double v = a[(i) * w + j];
            double v_old = b[(i) * w + j];
            double diff = fabs(v - v_old);
            sum += v;
            if (tmin > v) tmin = v;
            if (tmax < v) tmax = v;
            if (diff > maxdiff) maxdiff = diff;
        }

    r->niter = iter;
    r->maxdiff = maxdiff;
    r->tmin = tmin;
    r->tmax = tmax;
    r->tavg = sum / (p->N * p->M);

    r->time = (double)(after.tv_sec - before->tv_sec) + 
        (double)(after.tv_usec - before->tv_usec) / 1e6;

    return (maxdiff >= p->threshold) ? 0 : 1;
}

static const double c_cdir = 0.25 * M_SQRT2 / (M_SQRT2 + 1.0);
static const double c_cdiag = 0.25 / (M_SQRT2 + 1.0);

void __global__ do_calc(double *c, double *src, double *dst, size_t w, size_t h)
{
    unsigned x = blockIdx.x * blockDim.x + threadIdx.x + 1;
    unsigned y = blockIdx.y * blockDim.y + threadIdx.y + 1;
    if (x >= w-1)
        return;
    if (y >= h-1)
        return;

    double w_local = c[(y) * w + x];
    double restw = 1.0 - w_local;

    dst[(y) * w + x] = w_local * src[(y) * w + x] + 

	    (src[(y+1) * w + x  ] + src[(y-1) * w + x  ] + 
	     src[(y  ) * w + x+1] + src[(y  ) * w + x-1]) * (restw * c_cdir) +

	    (src[(y-1) * w + x-1] + src[(y-1) * w + x+1] + 
	     src[(y+1) * w + x-1] + src[(y+1) * w + x+1]) * (restw * c_cdiag);
}

void __global__ do_smear(double *dst, size_t w, size_t h) {
    unsigned y = blockIdx.x + 1;
    if (y >= h-1)
        return;

    dst[(y) * w + w-1] = dst[(y) * w + 1];
    dst[(y) * w + 0] = dst[(y) * w + w-2];
}

void do_compute(const struct parameters* p, struct results *r)
{
    size_t i, j;

    /* alias input parameters */
    const double *tinit = (const double *)p->tinit;
    const double *cinit = (const double *)p->conductivity;

    /* allocate grid data */
    const size_t h = p->N + 2;
    const size_t w = p->M + 2;
    double *g1 = (double *) malloc(h * w * sizeof(double));
    double *g2 = (double *) malloc(h * w * sizeof(double));

    /* allocate halo for conductivities */
    double *c = (double *) malloc(h * w * sizeof(double));

    /* set initial temperatures and conductivities */
    for (i = 1; i < h - 1; ++i)
        for (j = 1; j < w - 1; ++j) 
        {
            g1[(i) * w + j] = tinit[(i-1) * p->M + j-1];
            c[(i) * w + j] = cinit[(i-1) * p->M + j-1];
        }
    /* smear outermost columns to border */
    for (j = 1; j < w-1; ++j) {
        g1[(0) * w + j] = g2[(0) * w + j] = g1[(1) * w + j];
        g1[(h-1) * w + j] = g2[(h-1) * w + j] = g1[(h-2) * w + j];
    }
    /* smear outermost rows to borders */
    for (i = 0; i < h; ++i)
    {
        g1[(i) * w + w-1] = g2[(i) * w + w-1] = g1[(i) * w + 1];
        g1[(i) * w + 0] = g2[(i) * w + 0] = g1[(i) * w + w-2];
    }

    /* compute */
    size_t iter;
    double *src = g2;
    double *dst = g1;

    double *d_c, *d_dst, *d_src;
    CUDA_SAFE_CALL( hipMalloc(&d_src, sizeof(double) * w * h ) );
    CUDA_SAFE_CALL( hipMalloc(&d_dst, sizeof(double) * w * h ) );
    CUDA_SAFE_CALL( hipMalloc(&d_c, sizeof(double) * w * h ) );
    CUDA_SAFE_CALL( hipMemcpy(d_c, c, sizeof(double) * w * h, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy(d_dst, dst, sizeof(double) * w * h, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipMemcpy(d_src, src, sizeof(double) * w * h, hipMemcpyHostToDevice) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
	
    struct timeval before;
    gettimeofday(&before, NULL);

    for (iter = 1; iter <= p->maxiter; ++iter)
    {
        /* swap source and destination */
        { double *tmp = src; src = dst; dst = tmp; }
        { double *tmp = d_src; d_src = d_dst; d_dst = tmp; }

        const unsigned gridSize = 16;
        dim3 blockSize((h-2+gridSize-1)/gridSize, (w-2+gridSize-1)/gridSize);
        dim3 threadSize(gridSize, gridSize);
        do_calc<<<blockSize, threadSize>>>(d_c, d_src, d_dst, w, h);
        CUDA_SAFE_CALL( hipDeviceSynchronize() );
        do_smear<<<h-2, 1>>>(d_dst, w, h);
        CUDA_SAFE_CALL( hipDeviceSynchronize() );

        /* conditional reporting */
        if (iter % p->period == 0) {
            CUDA_SAFE_CALL( hipMemcpy(dst, d_dst, sizeof(double) * w * h, hipMemcpyDeviceToHost) );
            CUDA_SAFE_CALL( hipMemcpy(src, d_src, sizeof(double) * w * h, hipMemcpyDeviceToHost) );
            CUDA_SAFE_CALL( hipDeviceSynchronize() );
            if(fill_report(p, r, h, w, dst, src, iter, &before)) {iter++; break;}
            if(p->printreports) report_results(p, r);
        }
    }

    /* report at end in all cases */
    iter--;
    CUDA_SAFE_CALL( hipMemcpy(dst, d_dst, sizeof(double) * w * h, hipMemcpyDeviceToHost) );
    CUDA_SAFE_CALL( hipMemcpy(src, d_src, sizeof(double) * w * h, hipMemcpyDeviceToHost) );
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

    fill_report(p, r, h, w, dst, src, iter, &before);

    free(c);
    free(g2);
    free(g1);
}
